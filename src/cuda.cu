#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>

#include <assert.h>

#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"
#include "./headers/checkpoint.h"


int block_size = 1024;


int n_body;
int n_iteration;

__device__ __managed__ int bound_x_d = 4000;
__device__ __managed__ int bound_y_d = 4000;
__device__ __managed__ int max_mass_d = 400;
__device__ __managed__ double error_d = 1e-5f;
__device__ __managed__ double dt_d = 0.0001f;
__device__ __managed__ double gravity_const_d = 1000000.0f;
__device__ __managed__ double radius2_d = 0.01f;


__global__ void update_position(double *x, double *y, double *vx, double *vy, int n) {
    // update position 
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        x[i] += vx[i] * dt_d;
        y[i] += vy[i] * dt_d;
    }
}

__global__ void update_velocity(double *m, double *x, double *y, double *vx, double *vy, int n) {
    //TODO: calculate force and acceleration, update velocity
    int i = blockDim.x * blockIdx.x + threadIdx.x;  // update the ith element in the array
    if (i < n) {
        for (int j = 0; j < n; j++)
        {
            if (i == j) continue;

            // for each pair of bodies
            // calculate distance
            double distance_x = x[i] - x[j];
            double distance_y = y[i] - y[j];
            double distance = sqrt(distance_x * distance_x + distance_y * distance_y);

            
            // calculate force
            double force = gravity_const_d * m[i] * m[j] / (distance * distance + error_d); // the force scalar
            if (force > 35000) force = 35000;
            // calculate acceleration from j to i
            double acceleration_i = force / m[i];
            double acceleration_x_i = -acceleration_i * distance_x / distance;  // acceleration on i on x axis
            double acceleration_y_i = -acceleration_i * distance_y / distance;

            if (distance*distance < radius2_d)
            {
                // if the distance is too small, we will reverse the velocity of the two bodies
                vx[i] = -vx[i];
                vy[i] = -vy[i];
                break;
            }

            // update velocity
            vx[i] += acceleration_x_i * dt_d;
            vy[i] += acceleration_y_i * dt_d;
        }
        
    }
}

__global__ void check_bounds(double *x, double *y, double *vx, double *vy, int n_body) {
    //check if the body will go out of bounds. If so, it will bounce back
    for (int i = 0; i < n_body; i++)
    {
        if (x[i] <= 0 || x[i] >= bound_x_d)
        {
            vx[i] = -vx[i];
        }
        if (y[i] <= 0 || y[i] >= bound_y_d)
        {
            vy[i] = -vy[i];
        }
    }

} 



void generate_data(double *m, double *x,double *y,double *vx,double *vy, int n) {
    // TODO: Generate proper initial position and mass for better visualization
    srand((unsigned)time(NULL));
    for (int i = 0; i < n; i++) {
        m[i] = rand() % max_mass + 1.0f;
        x[i] = 2000.0f + rand() % (bound_x / 4);
        y[i] = 2000.0f + rand() % (bound_y / 4);
        vx[i] = 0.0f;
        vy[i] = 0.0f;
    }
}


void master() {
    double* m = new double[n_body];
    double* x = new double[n_body];
    double* y = new double[n_body];
    double* vx = new double[n_body];
    double* vy = new double[n_body];

    generate_data(m, x, y, vx, vy, n_body);

    Logger l = Logger("cuda", n_body, bound_x, bound_y);

    double *device_m;
    double *device_x;
    double *device_y;
    double *device_vx;
    double *device_vy;


    hipMalloc(&device_m, n_body * sizeof(double));
    hipMalloc(&device_x, n_body * sizeof(double));
    hipMalloc(&device_y, n_body * sizeof(double));
    hipMalloc(&device_vx, n_body * sizeof(double));
    hipMalloc(&device_vy, n_body * sizeof(double));

    hipMemcpy(device_m, m, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_x, x, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_y, y, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vx, vx, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vy, vy, n_body * sizeof(double), hipMemcpyHostToDevice);

    int n_block = n_body / block_size + 1; 

    for (int i = 0; i < n_iteration; i++){
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();


        update_velocity<<<n_block, block_size>>>(device_m, device_x, device_y, device_vx, device_vy, n_body);
        update_position<<<n_block, block_size>>>(device_x, device_y, device_vx, device_vy, n_body);
        check_bounds<<<1,1>>>(device_x, device_y, device_vx, device_vy, n_body);


        hipMemcpy(x, device_x, n_body * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(y, device_y, n_body * sizeof(double), hipMemcpyDeviceToHost);

        l.save_frame(x, y);

        std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> time_span = t2 - t1;
        printf("Iteration %d, elapsed time: %.3f\n", i, time_span);

        #ifdef GUI
        glClear(GL_COLOR_BUFFER_BIT);
        glColor3f(1.0f, 0.0f, 0.0f);
        glPointSize(2.0f);
        glBegin(GL_POINTS);
        double xi;
        double yi;
        for (int i = 0; i < n_body; i++){
            xi = x[i];
            yi = y[i];
            glVertex2f(xi, yi);
        }
        glEnd();
        glFlush();
        glutSwapBuffers();
        #else

        #endif

    }

    hipFree(device_m);
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_vx);
    hipFree(device_vy);

    hipFree(device_m);
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_vx);
    hipFree(device_vy);

    delete[] m;
    delete[] x;
    delete[] y;
    delete[] vx;
    delete[] vy;
    
}


int main(int argc, char *argv[]){
    
    n_body = atoi(argv[1]);
    n_iteration = atoi(argv[2]);

    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(500, 500);
    glutCreateWindow("N Body Simulation CUDA Implementation");
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    gluOrtho2D(0, bound_x, 0, bound_y);
    #endif

    master();

    printf("Student ID: 120090727\n"); // replace it with your student id
    printf("Name: Li Jiaqi\n"); // replace it with your name
    printf("Assignment 2: N Body Simulation CUDA Implementation\n");

    return 0;

}


